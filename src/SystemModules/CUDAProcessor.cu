#include "hip/hip_runtime.h"
#include "CUDAProcessor.h"
#include "../Math/IntVec2.h"
#include "../Profiling/Profiler.h"
#include <omp.h>
#include <new>
#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <>

using namespace std;
using namespace Graphics;

__device__ static int wrap(signedSize_t x, signedSize_t mod)
{
	if (x >= 0)
	{
		return x % mod;
	}
	//Otherwise, take modulus of x and add that to mod to get our result.
	return mod + (x % mod);
}

__device__ static void followSeam(IntVec2* outVec, SeamTracebackDirection direction, bool transpose)
{
	IntVec2 seamUp = IntVec2(!transpose ? 0 : -1, !transpose ? -1 : 0);
	IntVec2 seamLeft = IntVec2(!transpose ? -1 : 0, !transpose ? 0 : -1);
	switch (direction)
	{
	case SEAM_TB_LDOWN:
		*outVec += -seamUp + seamLeft;
		break;
	case SEAM_TB_LEFT:
		*outVec += seamLeft;
		break;
	case SEAM_TB_LUP:
		*outVec += seamUp + seamLeft;
		break;
	default:
		assert(false && "CUDAProcessor::removeSeam(): Seam data is corrupt!");
		abort();
		break;
	}
}

__device__ void CUDAProcessor::calcScharrAtPixel(signedSize_t x, signedSize_t y)
{
	//Unclear - can we filter in place?

	//Whatever the case, get our current pixel.
	auto pixel = energy.WritablePixelAt(x, y);
	//Now get the neighboring pixels.
	//Remember to wrap around if the pixel would be out of bounds.
	const int kNumPixels = 8;
	//Pixels are in order: up-left, up, up-right, right, down-right, down, down-left, left.
	//The filters are in the same order.
	Vec3 neighbors[] = {
	*image.PixelAt(wrap(x - 1, image.Width()),
		wrap(y - 1, image.Height())),
	*image.PixelAt(x,
		wrap(y - 1, image.Height())),
	*image.PixelAt(wrap(x + 1, image.Width()),
		wrap(y - 1, image.Height())),
	*image.PixelAt(wrap(x + 1, image.Width()),
		y),
	*image.PixelAt(wrap(x + 1, image.Width()),
		wrap(y + 1,
		image.Height())),
	*image.PixelAt(x,
		wrap(y + 1, image.Height())),
	*image.PixelAt(wrap(x - 1, image.Width()),
		wrap(y + 1, image.Height())),
	*image.PixelAt(wrap(x - 1, image.Width()),
		y)
	};
	const EnergyT ScharrX[] = {3, 0, -3, -10, -3, 0, 3, 10};
	const EnergyT ScharrY[] = {3, 10, 3, 0, -3, -10, -3, 0};

	//Now our horizontal gradient is the x filter applied to the neighbors;
	Vec3 horizGradientSquared = Vec3::Zero();
	for (int i = 0; i < kNumPixels; ++i)
	{
		horizGradientSquared += neighbors[i] * ScharrX[i];
	}
	//vertical is the y filter to the neighbors.
	Vec3 vertGradientSquared = Vec3::Zero();
	for (int i = 0; i < kNumPixels; ++i)
	{
		vertGradientSquared += neighbors[i] * ScharrY[i];
	}

	//Once again, take square root to it.
	*pixel = sqrtf(horizGradientSquared.SqrMag() + vertGradientSquared.SqrMag());
}

__global__ void CUDAProcessor::calcAllEnergy()
{
	//If this is out of bounds, abort.
	size_t pixelIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (pixelIdx >= imageNumPixels) {
		return;
	}

	//Get the energy at this block's pixel.
	int y = pixelIdx / imageWidth;
	int x = pixelIdx % imageWidth;
	calcScharrAtPixel(x, y);
}

__device__ void CUDAProcessor::recalcSeamEnergy(size_t seamIdx, bool transpose)
{
	//TODO: parallelize this.

	//This is before the seam data is recalculated, so we still have the seam path.
	//The image has had the seam removed, so we have the right intensities.

	//Figure out constants given transpose mode:
	//For moving along seam.
	IntVec2 seamUp = IntVec2(!transpose ? 0 : -1, !transpose ? -1 : 0);
	IntVec2 seamLeft = IntVec2(!transpose ? -1 : 0, !transpose ? 0 : -1);

	//Turn the index into actual X/Y coordinates.
	IntVec2 seamPos = IntVec2(!transpose ? image.Width() - 1 : 0,
		!transpose ? 0 : image.Height() - 1);
	seamPos += (-seamUp)*seamIdx;

	const SeamTracebackElem* currSeamElem = seamTraceback.PixelAt(seamPos);
	//For each pixel P in the seam:
	while (currSeamElem->TracebackDirection != SEAM_TB_END)
	{
		//recalc energy(xS, yS) - this was the lower/right side of the seam.
		//recalc energy of neighboring pixel (xS, yS-1) - this was the upper/left side of the seam.
		IntVec2 neighbor = seamPos + seamUp;
		//Neighbor may be out of bounds, wrap it.
		//neighbor.X() = wrap(neighbor.X(), energy.Width());
		//neighbor.Y() = wrap(neighbor.Y(), energy.Height());

		calcScharrAtPixel(seamPos.X(), seamPos.Y());
		if (energy.IsInBounds(neighbor))
		{
			calcScharrAtPixel(neighbor.X(), neighbor.Y());
		}
		//Get the next pixel on the seam.
		followSeam(&seamPos, currSeamElem->TracebackDirection, transpose);

		if (!energy.IsInBounds(seamPos)) {
			break;
		}

		currSeamElem = seamTraceback.PixelAt(seamPos);
	}
}

__global__ void CUDAProcessor::calcSeamCosts(bool transpose)
{
	IntVec2 pixelPos = IntVec2::Zero();

	//Figure out constants given transpose mode:
	//For moving along seam.
	const IntVec2 seamUp = IntVec2(!transpose ? 0 : -1, !transpose ? -1 : 0);
	const IntVec2 seamRight = IntVec2(!transpose ? 1 : 0, !transpose ? 0 : 1);

	const size_t bottomEdge = !transpose ? seamTraceback.Height() : seamTraceback.Width();
	const size_t rightEdge = !transpose ? seamTraceback.Width() : seamTraceback.Height();

	size_t pixelIdx = (blockIdx.x * blockDim.x) + threadIdx.x;

	//Quit if this pixel is out of bounds.
	if (pixelIdx >= bottomEdge) {
		return;
	}

	//Solve our pixel!
	pixelPos = (-seamUp)*pixelIdx;
	auto pixel = seamTraceback.WritablePixelAt(pixelPos);
	pixel->SeamCost = *energy.PixelAt(pixelPos);
	pixel->TracebackDirection = SEAM_TB_END;
}

__global__ void CUDAProcessor::calcSeamCosts(bool transpose)
{
	IntVec2 pixelPos = IntVec2::Zero();

	//Figure out constants given transpose mode:
	//For moving along seam.
	const IntVec2 seamUp = IntVec2(!transpose ? 0 : -1, !transpose ? -1 : 0);
	const IntVec2 seamRight = IntVec2(!transpose ? 1 : 0, !transpose ? 0 : 1);

	const size_t bottomEdge = !transpose ? seamTraceback.Height() : seamTraceback.Width();
	const size_t rightEdge = !transpose ? seamTraceback.Width() : seamTraceback.Height();

	size_t pixelIdx = (blockIdx.x * blockDim.x) + threadIdx.x;

	//Quit if this pixel is out of bounds.
	if (pixelIdx >= bottomEdge) {
		return;
	}

	//Now solve the pixels ahead of them.
	//For each column (can't parallelize this):
	for (size_t i = 1; i < rightEdge; ++i)
	{
		//Solve our element of the row.
		pixelPos = seamRight*i - seamUp*pixelIdx;
		auto pixel = seamTraceback.WritablePixelAt(pixelPos);

		//The left seam cost will always be valid.
		IntVec2 leftPos = pixelPos - seamRight;
		IntVec2 upLeftPos = leftPos + seamUp;
		IntVec2 downLeftPos = leftPos - seamUp;
		EnergyT leftCost = seamTraceback.PixelAt(leftPos)->SeamCost;
		EnergyT upLeftCost = seamTraceback.IsInBounds(upLeftPos) ?
							seamTraceback.PixelAt(upLeftPos)->SeamCost : FLT_MAX;
		EnergyT downLeftCost = seamTraceback.IsInBounds(downLeftPos) ?
			seamTraceback.PixelAt(downLeftPos)->SeamCost : FLT_MAX;

		EnergyT minCost = fminf(fminf(leftCost, upLeftCost), downLeftCost);
			
		//Now set our pixel's cost.
		pixel->SeamCost = *energy.PixelAt(pixelPos) + minCost;
		//Figure out our pixel's traceback direction.
		if (minCost == leftCost)
		{
			pixel->TracebackDirection = SEAM_TB_LEFT;
		}
		else if (minCost == upLeftCost)
		{
			pixel->TracebackDirection = SEAM_TB_LUP;
		}
		else if (minCost == downLeftCost)
		{
			pixel->TracebackDirection = SEAM_TB_LDOWN;
		}
	}
}

__device__ size_t CUDAProcessor::findMinCostSeam(bool transpose)
{
	IntVec2 currPos = IntVec2(!transpose ? image.Width() - 1 : 0,
		!transpose ? 0 : image.Height() - 1);
	//Scan right side of image.
	IntVec2 incr = IntVec2(!transpose ? 0 : 1,
		!transpose ? 1 : 0);
	const size_t scanLen = !transpose ? image.Height() : image.Width();
	size_t minIdx = 0;
	EnergyT minSeamCost = FLT_MAX;

	//TODO: Is this parallelizable under the server's OpenMP version?
	//For each pixel at the edge:
	for (size_t i = 0; i < scanLen; ++i)
	{
		auto currPixel = seamTraceback.PixelAt(currPos + incr*i);
		//Is this cost lower than the last known cost?
		if (currPixel->SeamCost < minSeamCost)
		{
			//If so, update last observed minimum.
			minSeamCost = currPixel->SeamCost;
			minIdx = i;
		}
	}

	//Return the seam.
	return minIdx;
}

__device__ CUDAProcessor::SeamRemoveDirection CUDAProcessor::removeSeam(size_t seamIdx, bool transpose)
{
	SeamRemoveDirection removeDirection = REMOVE_DIRECTION_DOWN;
	//Figure out constants given transpose mode:
	//For getting seam start.
	//For moving along seam.
	IntVec2 seamUp = IntVec2(!transpose ? 0 : -1, !transpose ? -1 : 0);
	IntVec2 seamLeft = IntVec2(!transpose ? -1 : 0, !transpose ? 0 : -1);

	//Each pixel in the seam gets removed.
	//Turn the index into actual X/Y coordinates.
	IntVec2 seamPos = IntVec2(!transpose ? image.Width() - 1 : 0,
		!transpose ? 0 : image.Height() - 1);
	seamPos += (-seamUp)*seamIdx;

	const SeamTracebackElem* currSeamElem = seamTraceback.PixelAt(seamPos);
	//Which way should we remove?
	size_t midpoint = (transpose ? image.Height() : image.Width()) / 2;
	removeDirection = seamIdx < midpoint ? REMOVE_DIRECTION_UP : REMOVE_DIRECTION_DOWN;
	ptrdiff_t directionFactor = removeDirection == REMOVE_DIRECTION_UP ? 1 : -1;

	//TODO: parallelize this.
	//In order to do this, we need the position of the pixel on the seam
	//WITHOUT following the seam. This means storing an extra buffer and removing the traceback code.
	//May not be worth doing.

	//For each pixel P in the seam:
	while (currSeamElem->TracebackDirection != SEAM_TB_END)
	{
		//For each pixel P' below P: (or to the right, if this is a vertical seam)
		IntVec2 pixelToCopyPos = seamPos;
		IntVec2 nextPixelPos = pixelToCopyPos + seamUp*directionFactor;
		while (image.IsInBounds(pixelToCopyPos) && image.IsInBounds(nextPixelPos))
		{
			//P.value = P'.value
			image.SetPixelAt(pixelToCopyPos, *image.PixelAt(nextPixelPos));
			//Copy the energy too!
			energy.SetPixelAt(pixelToCopyPos, *energy.PixelAt(pixelToCopyPos));

			//Move to the next pixel.
			pixelToCopyPos = nextPixelPos;
			nextPixelPos = pixelToCopyPos + seamUp*directionFactor;
		}

		//Get the next pixel on the seam.
		followSeam(&seamPos, currSeamElem->TracebackDirection, transpose);

		//ugh: early out if we somehow went out of bounds.
		if (!image.IsInBounds(seamPos)) {
			break;
		}

		currSeamElem = seamTraceback.PixelAt(seamPos);
	}

	return removeDirection;
}

__global__ void CUDAProcessor::updateBounds(size_t oppositeRowsLeft, SeamRemoveDirection seamRemoveDirection) {
	IntVec2 removeVec = IntVec2(0, 0);
	if (removeMode == REMOVE_ROWS)
	{
		removeVec = IntVec2(0, 1);
	}
	else
	{
		removeVec = IntVec2(1, 0);
	}
	if (seamRemoveDirection == REMOVE_DIRECTION_DOWN)
	{
		image.SetEnd(image.End() - removeVec);
		energy.SetEnd(energy.End() - removeVec);
		seamTraceback.SetEnd(seamTraceback.End() - removeVec);
	}
	else
	{
		image.SetOrigin(image.Origin() + removeVec);
		energy.SetOrigin(energy.Origin() + removeVec);
		seamTraceback.SetOrigin(seamTraceback.Origin() + removeVec);
	}

	//Check the other count - do we need to transpose?
	RemoveMode oppositeMode = removeMode == REMOVE_ROWS ? REMOVE_COLS : REMOVE_ROWS;//(RemoveMode)((removeMode + 1) % REMOVE_MODE_COUNT);
	if (oppositeRowsLeft > 0)
	{
		//If so, transpose at this point.
		removeMode = oppositeMode;
	}
}

void CUDAProcessor::doProcessImage(size_t numRowsToRemove, size_t numColsToRemove, size_t numCores)
{
	//Note how many threads we'll be using.
	std::printf("CUDAProcessor::ProcessImage(): Using %d cores\n", numCores);

	//Calculate the initial energy gradient of the image.
	profiler.StartProfile(ProfileCode::PC_CALC_ALL_ENERGY);
	//X*Y operation; split evenly among SMs.
	size_t numThreads = imageNumPixels / numCores;
	assert(numThreads*numCores == imageNumPixels && "Cores don't evenly split image pixels!");
	calcAllEnergy<<<numCores, numThreads>>>(numThreads);
	profiler.EndProfile(ProfileCode::PC_CALC_ALL_ENERGY);

#if defined(_DEBUG)
	{
		//Write the energy to output.
		EnergyBuffer tempEnergy(imageWidth, imageHeight);
		energy.CopyToHostBuffer(tempEnergy);
		WriteEnergyBuffer(tempEnergy, "energyBuffer.bmp");
	}
#endif

	//How many rows to remove?
	//How many columns to remove?
	size_t rowsColsToRemove[2] = { numRowsToRemove, numColsToRemove };

	//Start by removing rows.
	RemoveMode removeMode = numRowsToRemove > 0 ? REMOVE_ROWS : REMOVE_COLS;

	std::printf("CUDAProcessor::ProcessImage(): Removing %d rows and %d columns\n", rowsColsToRemove[REMOVE_ROWS], rowsColsToRemove[REMOVE_COLS]);
	//While you have too many rows/columns:
	while (rowsColsToRemove[REMOVE_ROWS] > 0 || rowsColsToRemove[REMOVE_COLS] > 0)
	{
		bool transpose = removeMode != REMOVE_ROWS;

		//Find the cost of each seam in the image.
		profiler.StartProfile(ProfileCode::PC_CALC_SEAM_COSTS);
		//Y operation.
		size_t numThreads = !transpose ? imageHeight : imageWidth;
		numThreads /= numCores;
		resetSeamStart<<<numCores, numThreads>>>(transpose, numThreads);
		calcSeamCosts<<<numCores, numThreads>>>(transpose, numThreads);
		profiler.EndProfile(ProfileCode::PC_CALC_SEAM_COSTS);

		//Remove minimum cost seam.
		profiler.StartProfile(ProfileCode::PC_FIND_MIN_COST_SEAM);
		TODO;
		//Unclear if parallelizable, but is a Y operation.
		size_t targetSeamIdx = findMinCostSeam(transpose);
		profiler.EndProfile(ProfileCode::PC_FIND_MIN_COST_SEAM);
		profiler.StartProfile(ProfileCode::PC_REMOVE_SEAM);
		//Another part that may not be parallelizable. X operation.
		TODO;
		auto seamRemoveDirection = removeSeam(targetSeamIdx, transpose);
		profiler.EndProfile(ProfileCode::PC_FIND_MIN_COST_SEAM);

		//Now the image has been modified; recalculate the energy near the removed seam.
		profiler.StartProfile(ProfileCode::PC_RECALC_SEAM_ENERGY);
		//Hasn't been parallelized. Unknown dimensions.
		recalcSeamEnergy(targetSeamIdx, transpose);
		profiler.EndProfile(ProfileCode::PC_RECALC_SEAM_ENERGY);
		//Seam traceback doesn't have to be updated since
		//calcSeamCosts() must evaluate the entire image.

		//Update buffer dimensions.
		profiler.StartProfile(ProfileCode::PC_BOUNDS_ADJUST);
		rowsColsToRemove[removeMode] -= 1;
		updateBounds<<<1, 1>>>(rowsColsToRemove[oppositeMode], seamRemoveDirection);
		profiler.EndProfile(ProfileCode::PC_BOUNDS_ADJUST);
	}
}

CUDAProcessor::CUDAProcessor(LABColorBuffer& pImage, Profiler& pProfiler) : image(pImage.Width(), pImage.Height()), energy(pImage.Width(), pImage.Height()), seamTraceback(pImage.Width(), pImage.Height()), profiler(pProfiler)
{
	imageWidth = pImage.Width();
	imageHeight = pImage.Height();
	imageNumPixels = imageWidth*imageHeight;
	//Copy source image to device.
	image.CopyFromHostBuffer(pImage);
	removeMode = REMOVE_ROWS;
}

CUDAProcessor::~CUDAProcessor()
{
}

void CUDAProcessor::ProcessImage(size_t numRowsToRemove, size_t numColsToRemove)
{
	//Do the actual work.
	//Should probably lookup how many blocks
	//the device has available first.
	doProcessImage(numRowsToRemove, numColsToRemove, numCores);
}